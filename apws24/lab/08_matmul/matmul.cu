#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;


__global__ void matmul_kernel(float *_A, float *_B, float *_C, int M, int N, int K) {



  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= M || j >= N) return;

  float sum = 0.;

  for (int n = 0; n < K; n++)
  {
    sum += _A[i * K + n] * _B[n * N + j];
  }

  _C[i * N + j] = sum;

}


// void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
//   for (int i = 0; i < M; i++) {
//     for (int k = 0; k < K; k++) {
//       for (int j = 0; j < N; j++) {
//         _C[i * N + j] += _A[i * K + k] * _B[k * N + j];
//       }
//     }
//   }
// }

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // Remove this line after you complete the matmul on GPU
  //naive_cpu_matmul(_A, _B, _C, M, N, K);


  // (TODO) Upload A and B matrix to GPU

  hipMemcpy(A_gpu, _A, sizeof(float)*M*K, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, _B, sizeof(float)*K*N, hipMemcpyHostToDevice);
  
  // (TODO) Launch kernel on a GPU

  
  dim3 blockdim(32, 32);
  dim3 gridDim((M + blockdim.x - 1) / blockdim.x,
               (N + blockdim.y - 1) / blockdim.y);

  matmul_kernel<<<gridDim, blockdim>>>(A_gpu, B_gpu, C_gpu,
                                            M, N, K);

  // (TODO) Download C matrix from GPU
  CHECK_CUDA(hipMemcpy(_C, C_gpu,  
                sizeof(float)*M*N, hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory

  
  CHECK_CUDA(hipMalloc(&A_gpu, sizeof(float) * M * K));
  CHECK_CUDA(hipMalloc(&B_gpu, sizeof(float) * K * N));
  CHECK_CUDA(hipMalloc(&C_gpu, sizeof(float) * M * N));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.



  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
