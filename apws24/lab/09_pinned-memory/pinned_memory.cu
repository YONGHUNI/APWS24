
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  int bytes = sizeof(int) * (1 << 20);

  int *d_a;
  CHECK_CUDA(hipMalloc(&d_a, bytes));

  /* 1. Pageable memory test */
  {
    int *a_pageable;
    // TODO: Allocate pageable memory using malloc
    a_pageable = (int *) malloc(bytes);

    auto start = std::chrono::system_clock::now();
    // TODO: Run H2D memcpy on pageable memory
    CHECK_CUDA(hipMemcpy(d_a, a_pageable, sizeof(bytes), hipMemcpyHostToDevice));
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = end - start;
    printf("Pageable memory bandwidth: %lf GB/s\n",
           (bytes / diff.count() / 1e9));
           
    free(a_pageable);
  }

  /* 2. Pinned memory test */
  {
    int *a_pinned;
    // TODO: Allocate pinned memory using cudaMallocHost
    CHECK_CUDA(hipHostMalloc(&a_pinned, bytes, hipHostMallocDefault));

    auto start = std::chrono::system_clock::now();
    // TODO: Run H2D memcpy on pinned memory
    CHECK_CUDA(hipMemcpy(d_a, a_pinned, sizeof(bytes), hipMemcpyHostToDevice));
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = end - start;
    printf("Pinned memory bandwidth: %lf GB/s\n",
           (bytes / diff.count() / 1e9));
  }

  CHECK_CUDA(hipMalloc(&d_a, bytes));


  return 0;
}
