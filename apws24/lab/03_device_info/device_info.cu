
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  int count;
  CHECK_CUDA(hipGetDeviceCount(&count));

  printf("Number of devices: %d\n", count);
  hipDeviceProp_t props[4];
  for (int i = 0; i < count; ++i) {
    printf("\tdevice %d:\n", i);

    // TODO: get and print device properties
    hipGetDeviceProperties(&props[i], i);

    printf("\t\tName: %s\n",props[i].name);
    printf("\t\tSM Count: %d\n",props[i].multiProcessorCount);
    printf("\t\tMax Thread per Block: %d\n",props[i].maxThreadsPerBlock);
    printf("\t\tTotal Global Memory: %zd\n",props[i].totalGlobalMem);
    printf("\t\tTotal Shared Memory per Block: %zd\n",props[i].sharedMemPerBlock);

    // END TODO
  }

  return 0;
}
